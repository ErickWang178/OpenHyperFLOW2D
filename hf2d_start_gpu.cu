
#include <hip/hip_runtime.h>
/*******************************************************************************
*   OpenHyperFLOW2D-CUDA                                                       *
*                                                                              *
*   Version  2.0.1                                                             *
*   Copyright (C)  1995-2016 by Serge A. Suchkov                               *
*   Copyright policy: LGPL V3                                                  *
*   http://github.com/sergeas67/openhyperflow2d                                *
*                                                                              *
*   hf2d_start_gpu.cpp: OpenHyperFLOW2D-CUDA solver init code....              *
*                                                                              *
*  last update: 14/04/2016                                                     *
********************************************************************************/
#ifdef _CUDA_
#define _PARALLEL_ONLY

#include <cuda.h>
#include <cuda_runtime.h>
#include <cuda_runtime_api.h>

#include <sys/time.h>
#include <sys/timeb.h>
#include "libDEEPS2D/deeps2d_core.hpp"
 
SolverMode ProblemType;

int rank;
int last_rank;
int warp_size = 0;
FP _dt_test=1.0;

timeval mark1, mark2;
cudaDeviceProp dprop;

// Arrays for multiGPU
UArray< FlowNode2D<FP,NUM_COMPONENTS>* >*     cudaArraySubDomain      = NULL;
UArray< FlowNodeCore2D<FP,NUM_COMPONENTS>* >* cudaArrayCoreSubDomain  = NULL;
UArray< XY<int>  >*                           cudaDimArray            = NULL;
UArray< XY<int>* >*                           cudaWallNodesArray      = NULL;
UArray< ChemicalReactionsModelData2D* >*      cudaCRM2DArray          = NULL;
XY<int>*                                      cudaWallNodes           = NULL;
FlowNode2D<FP,NUM_COMPONENTS>*                cudaSubDomain           = NULL;
FlowNodeCore2D<FP,NUM_COMPONENTS>*            cudaCoreSubDomain       = NULL;
ChemicalReactionsModelData2D*                 cudaCRM2D               = NULL;
UArray< MonitorPoint >*                       MonitorPointsArray      = NULL;
FP*                                           cudaHu;
UArray<FP*>*                                  cudaHuArray;
FP                                            x0;
FP                                            dx;
FP                                            dy;

int num_gpus = 0;   // number of CUDA GPUs
int max_num_threads = 0;
size_t max_gpu_memsize = 0;

size_t task_size;
cudaError_t cudaState;

UArray< FP >*                                 WallNodesUw_2D = NULL;
int                                           NumWallNodes;
int                                           isSingleGPU = 0;
int                                           ActiveSingleGPU = 0;
int                                           ThreadBlockSize = 0; // Default = 0 ... doing calibration

int start_OpenHyperFLOW2D_GPU( int argc, char **argv )
{
    char                           inFile[256];
    ChemicalReactionsModelData2D   TmpCRM2D;

    FlowNode2D<FP,NUM_COMPONENTS>* TmpMatrixPtr;
    //FlowNodeCore2D<FP,NUM_COMPONENTS>* TmpCoreMatrixPtr;

    int TmpMaxX;

    ostream*     o_stream = &cout;

#ifdef _DEBUG_0
    ___try {
#endif // _DEBUG_0
            sprintf(inFile,"%s",argv[1]);
            Data = new InputData(inFile,DS_FILE,o_stream,0,10);
            if (Data->GetDataError()!=0) {
                *o_stream << "\nInput data error.\n" ;
                 o_stream->flush();
                 Exit_OpenHyperFLOW2D(0);
            }
        printf("\n\n\t");
        cudaGetDeviceCount(&num_gpus);

        if (num_gpus < 1) {
         printf("no CUDA capable devices were detected\n");
         Exit_OpenHyperFLOW2D(0);
        }

        printf("Number of CUDA devices:\t%d\n", num_gpus);

        for (int i = 0; i < num_gpus; i++)
        {
         cudaGetDeviceProperties(&dprop, i);
         printf("\t   %d: %s\n", i, dprop.name);
         max_num_threads  = dprop.maxThreadsPerBlock;
         printf("\t   Max threads per block: %d\n", max_num_threads);
         max_gpu_memsize = dprop.totalGlobalMem;
         printf("\t   Max GPU memory size: %lu\n", max_gpu_memsize);
         printf("\t   Number of  multiprocessors: %d\n",dprop.multiProcessorCount);
         printf("\t   Is kernels concurrent: %d\n",dprop.concurrentKernels);
         warp_size = dprop.warpSize;
         printf("\t   Warp size: %d\n",warp_size);
         printf("\t   Enable timeout: %i\n\n",dprop.kernelExecTimeoutEnabled);
        }

        // Init shared data
        InitSharedData(Data,&chemical_reactions);

        if(isSingleGPU)
           num_threads = num_blocks = num_gpus = 1;
        else
           num_threads = num_blocks = num_gpus;

        if(num_gpus > 1) {
           printf("Using multi GPU mode.\n\n");
        } else {
           printf("Using single GPU mode.\nActive GPU No:%i\n",ActiveSingleGPU);

        }

       //Create arrays  
        cudaArraySubDomain      =  new UArray< FlowNode2D<FP,NUM_COMPONENTS>* >();
        cudaArrayCoreSubDomain  =  new UArray< FlowNodeCore2D<FP,NUM_COMPONENTS>* >();
        cudaDimArray            =  new UArray< XY<int> >();
        cudaHuArray             =  new UArray< FP* >();
        dt_min_host_Array       =  new UArray<unsigned int*>();
        dt_min_device_Array     =  new UArray<unsigned int*>();
        cudaWallNodesArray      =  new UArray< XY<int>* >();
        cudaCRM2DArray          =  new UArray< ChemicalReactionsModelData2D* >();

        // Init solver (run on host)
        InitDEEPS2D((void*)o_stream);

        if(ProblemType == SM_NS) {
            *o_stream << "\nSolver Mode: Navier-Stokes/FP" << 8*sizeof(FP) <<"\n" << endl;
        } else {
            *o_stream << "\nSolver Mode: Euler/FP" << 8*sizeof(FP) <<"\n" << endl;
        }

        cudaStream_t *cuda_streams = (cudaStream_t *) malloc(num_gpus * sizeof(cudaStream_t));
        cudaEvent_t  *cuda_events = (cudaEvent_t *) malloc(num_gpus * sizeof(cudaEvent_t));

        for (int i = 0; i < num_gpus; i++) {

            int i_gpu;
            if(isSingleGPU) {
                i_gpu = ActiveSingleGPU;
            } else {
                i_gpu = i;
            }

             if(cudaSetDevice(i_gpu) != cudaSuccess ) {
                *o_stream << "\nError set CUDA device no: "<< i_gpu << endl;
                Exit_OpenHyperFLOW2D(num_gpus);
             }

             cudaState = cudaStreamCreate(&(cuda_streams[i]));
             if(cudaState != cudaSuccess ) {
                *o_stream << "\nError create stream no: "<< i << endl;
                printf("%s\n", cudaGetErrorString( cudaGetLastError() ) );
                Exit_OpenHyperFLOW2D(num_gpus);
             }

             cudaState = cudaEventCreate(&(cuda_events[i]));
             if(cudaState != cudaSuccess ) {
                *o_stream << "\nError create event no: "<< i << endl;
                printf("%s\n", cudaGetErrorString( cudaGetLastError() ) );
                Exit_OpenHyperFLOW2D(num_gpus);
             }

             // Load components properties
             LoadTable2GPU(chemical_reactions.Cp_OX,   TmpCRM2D.Cp_OX,   i_gpu);
             LoadTable2GPU(chemical_reactions.Cp_Fuel, TmpCRM2D.Cp_Fuel, i_gpu);
             LoadTable2GPU(chemical_reactions.Cp_cp,   TmpCRM2D.Cp_cp,   i_gpu);
             LoadTable2GPU(chemical_reactions.Cp_air,  TmpCRM2D.Cp_air,  i_gpu);

             LoadTable2GPU(chemical_reactions.mu_air,  TmpCRM2D.mu_air,  i_gpu);
             LoadTable2GPU(chemical_reactions.mu_cp,   TmpCRM2D.mu_cp,   i_gpu);
             LoadTable2GPU(chemical_reactions.mu_Fuel, TmpCRM2D.mu_Fuel, i_gpu);
             LoadTable2GPU(chemical_reactions.mu_OX,   TmpCRM2D.mu_OX,   i_gpu);

             if(ProblemType == SM_NS) {
                 LoadTable2GPU(chemical_reactions.lam_air, TmpCRM2D.lam_air, i_gpu);
                 LoadTable2GPU(chemical_reactions.lam_cp,  TmpCRM2D.lam_cp,  i_gpu);
                 LoadTable2GPU(chemical_reactions.lam_Fuel,TmpCRM2D.lam_Fuel,i_gpu);
                 LoadTable2GPU(chemical_reactions.lam_OX,  TmpCRM2D.lam_OX,  i_gpu);
             }

             TmpCRM2D.H_air  = chemical_reactions.H_air;
             TmpCRM2D.H_cp   = chemical_reactions.H_cp;
             TmpCRM2D.H_Fuel = chemical_reactions.H_Fuel;
             TmpCRM2D.H_OX   = chemical_reactions.H_OX;

             TmpCRM2D.R_air  = chemical_reactions.R_air;
             TmpCRM2D.R_cp   = chemical_reactions.R_cp;
             TmpCRM2D.R_Fuel = chemical_reactions.R_Fuel;
             TmpCRM2D.R_OX   = chemical_reactions.R_OX;

             TmpCRM2D.K0     = chemical_reactions.K0;
             TmpCRM2D.Tf     = chemical_reactions.Tf;
             TmpCRM2D.gamma  = chemical_reactions.gamma;

             if(cudaMalloc( (void**)&cudaCRM2D, sizeof(ChemicalReactionsModelData2D) ) == cudaErrorMemoryAllocation) {
                *o_stream << "\nError allocate GPU memory for CRM2D on device no:" << i_gpu << endl;
                Exit_OpenHyperFLOW2D(num_gpus);
             }

             CopyHostToDevice(&TmpCRM2D,cudaCRM2D,sizeof(ChemicalReactionsModelData2D));

             cudaCRM2DArray->AddElement(&cudaCRM2D);

       }

        for(int i_dev=0; i_dev < num_gpus;i_dev++) {

            int i_gpu;
            if(isSingleGPU) {
                i_gpu = ActiveSingleGPU;
            } else {
                i_gpu = i_dev;
            }

#ifdef _P2P_ACCESS_
            if((num_gpus>1)&&(i_dev+1 < num_gpus)) {
                SetP2PAccess(i_dev,i_dev+1);
            }
#endif // _P2P_ACCESS_

            if(cudaSetDevice(i_gpu) != cudaSuccess ) {
               *o_stream << "\nError set CUDA device no: "<< i_gpu << endl;
               Exit_OpenHyperFLOW2D(num_gpus);
            }

            cudaState = cudaMalloc( (void**)&cudaHu, sizeof(FP)*(NUM_COMPONENTS+1) );
            if(cudaState == cudaErrorMemoryAllocation) {
               *o_stream << "\nError allocate GPU memory for Hu[]" << endl;
               Exit_OpenHyperFLOW2D(num_gpus);
            }

            CopyHostToDevice(FlowNode2D<FP,NUM_COMPONENTS>::Hu,cudaHu,sizeof(FP)*(NUM_COMPONENTS+1));

            cudaHuArray->AddElement(&cudaHu);

#ifdef _DEVICE_MMAP_
            cudaState = cudaHostAlloc( (void**)&dt_min_host, sizeof(unsigned int), cudaHostAllocMapped ); 

            dt_min_host_Array->AddElement(&dt_min_host);  

            if(cudaState == cudaErrorMemoryAllocation) {
               *o_stream << "\nError allocate GPU memory for dt_min" << endl;
               Exit_OpenHyperFLOW2D(num_gpus);
            }

            cudaState = cudaHostGetDevicePointer( &dt_min_device, dt_min_host, 0 );

            dt_min_device_Array->AddElement(&dt_min_device);

            if(cudaState == cudaErrorMemoryAllocation) {
               *o_stream << "\nError mapped GPU memory for dt_min" << endl;
               Exit_OpenHyperFLOW2D(num_gpus);
            }
#else
            cudaState = cudaMalloc( (void**)&dt_min_device, sizeof(unsigned int));

            dt_min_device_Array->AddElement(&dt_min_device);

            if(cudaState == cudaErrorMemoryAllocation) {
               *o_stream << "\nError allocate GPU memory for dt_min "<<  endl;
               Exit_OpenHyperFLOW2D(num_gpus);
            }
#endif // _DEVICE_MMAP_
        }

        if(ProblemType == SM_NS) {

            // Scan area for seek wall nodes
            WallNodes = GetWallNodes((ofstream*)o_stream,J,Data->GetIntVal((char*)"isVerboseOutput")); 

            NumWallNodes = WallNodes->GetNumElements();
            *o_stream << NumWallNodes << " wall nodes found" << endl; 
        }

        gettimeofday(&mark2,NULL);

        for (int i_dev=0; i_dev < num_gpus;i_dev++) {

            int i_gpu;
            if(isSingleGPU) {
                i_gpu = ActiveSingleGPU;
            } else {
                i_gpu = i_dev;
            }

            if(cudaSetDevice(i_gpu) != cudaSuccess ) {
               *o_stream << "\nError set CUDA device no: "<< i_gpu << endl;
               Exit_OpenHyperFLOW2D(num_gpus);
            }

            if(ProblemType == SM_NS && NumWallNodes > 0) {

                cudaState = cudaMalloc( (void**)&cudaWallNodes, sizeof(XY<int>)*NumWallNodes );
                if(cudaState == cudaErrorMemoryAllocation) {
                   *o_stream << "\nError allocate GPU memory for WallNodes array." << endl;
                   Exit_OpenHyperFLOW2D(num_gpus);
                }

                if(cudaState != cudaSuccess ) {
                   *o_stream << "\nError set CUDA device no: "<< i_dev << endl;
                   Exit_OpenHyperFLOW2D(num_gpus);
                }

                CopyHostToDevice(WallNodes->GetArrayPtr(),cudaWallNodes,sizeof(XY<int>)*NumWallNodes);

                cudaWallNodesArray->AddElement(&cudaWallNodes);
            }
        }

        TmpMatrixPtr=J->GetMatrixPtr();
        int SubStartIndex, SubMaxX,r_Overlap=0,l_Overlap=0;
        SubStartIndex = 0;
        int iX0=0;

        *o_stream << "Allocate SubDomain:\n";

        //Allocate GPU buffers

        for (unsigned int i=0;i<GlobalSubDomain->GetNumElements();i++) {

            int i_gpu;
            if(isSingleGPU) {
                i_gpu = ActiveSingleGPU;
            } else {
                i_gpu = i;
            }

            if(cudaSetDevice(i_gpu) != cudaSuccess ) {
               *o_stream << "\nError set CUDA device no: "<< i_gpu << endl;
               Exit_OpenHyperFLOW2D(num_gpus);
            }

            SubStartIndex = GlobalSubDomain->GetElementPtr(i)->GetX();  
            SubMaxX = GlobalSubDomain->GetElementPtr(i)->GetY();

            if(i == GlobalSubDomain->GetNumElements()-1)
              r_Overlap = 0;
            else
              r_Overlap = 1;
            if(i == 0)
              l_Overlap = 0;
            else
              l_Overlap = 1;

            TmpMaxX = (SubMaxX-SubStartIndex) + l_Overlap;

            // Allocate FlowNode2D<FP,NUM_COMPONENTS> subdomain
            cudaState = cudaMalloc( (void**)&cudaSubDomain, (sizeof(FlowNode2D<FP,NUM_COMPONENTS>))*(TmpMaxX*MaxY) );

            if(cudaState == cudaErrorMemoryAllocation) {
               *o_stream << "\nError allocate GPU memory for SubDomain"<< endl;
               Exit_OpenHyperFLOW2D(num_gpus);
            }

           cudaArraySubDomain->AddElement(&cudaSubDomain);

           // Allocate FlowNodeCore2D<FP,NUM_COMPONENTS> subdomain
           cudaState = cudaMalloc( (void**)&cudaCoreSubDomain, (sizeof(FlowNodeCore2D<FP,NUM_COMPONENTS>))*(TmpMaxX*MaxY) );

           if(cudaState == cudaErrorMemoryAllocation) {
              *o_stream << "\nError allocate GPU memory for CoreSubDomain"<< endl;
              Exit_OpenHyperFLOW2D(num_gpus);
           }
           cudaArrayCoreSubDomain->AddElement(&cudaCoreSubDomain);
        }

        for (unsigned int i=0;i<GlobalSubDomain->GetNumElements();i++) {

            XY<int> TmpDim;

            int i_gpu;
            if(isSingleGPU) {
                i_gpu = ActiveSingleGPU;
            } else {
                i_gpu = i;
            }

            if(cudaSetDevice(i_gpu) != cudaSuccess ) {
               *o_stream << "\nError set CUDA device no: "<< i_gpu << endl;
               Exit_OpenHyperFLOW2D(num_gpus);
            }

            if(i == GlobalSubDomain->GetNumElements()-1)
              r_Overlap = 0;
            else
              r_Overlap = 1;
            if(i == 0)
              l_Overlap = 0;
            else
              l_Overlap = 1;

            SubStartIndex = GlobalSubDomain->GetElementPtr(i)->GetX();
            SubMaxX = GlobalSubDomain->GetElementPtr(i)->GetY();

            TmpMaxX = (SubMaxX-SubStartIndex) - r_Overlap; 
            TmpMatrixPtr = (FlowNode2D<FP,NUM_COMPONENTS>*)((ulong)J->GetMatrixPtr()+(ulong)(sizeof(FlowNode2D<FP,NUM_COMPONENTS>)*(SubStartIndex)*MaxY));

            int num_cuda_threads = warp_size;
            int num_cuda_blocks  = (TmpMaxX*MaxY)/num_cuda_threads;

            if (num_cuda_blocks*num_cuda_threads != TmpMaxX*MaxY)
                num_cuda_blocks++;

            x0 = SubStartIndex*FlowNode2D<FP,NUM_COMPONENTS>::dx;

            *o_stream << "SubDomain("<<i<<")[" << TmpMaxX << "x" << MaxY << "]  Size=" << (ulong)(sizeof(FlowNode2D<FP,NUM_COMPONENTS>)*TmpMaxX*MaxY)/(1024*1024) << " Mb\n"; 

            cudaSubDomain = cudaArraySubDomain->GetElement(i);

            CopyHostToDevice(TmpMatrixPtr,cudaSubDomain,(sizeof(FlowNode2D<FP,NUM_COMPONENTS>))*(TmpMaxX*MaxY));

            cudaHu =  cudaHuArray->GetElement(i);

            if(NumWallNodes > 0 &&
               ProblemType == SM_NS ) {

                cudaWallNodes = cudaWallNodesArray->GetElement(i);

                *o_stream << "GPU no: " << i_gpu << endl; 
                *o_stream << "CUDA threads: " << num_cuda_threads << endl;
                *o_stream << "CUDA thread blocks : " << num_cuda_blocks << endl;

                *o_stream << "\nParallel calc min distance to wall..." << endl;
                *o_stream << "Run cuda_SetMinDistanceToWall2D kernel..." << flush;

                cuda_SetMinDistanceToWall2D<<<num_cuda_blocks,num_cuda_threads, 0, cuda_streams[i]>>>(cudaSubDomain,
                                                                                                      TmpMaxX*MaxY,
                                                                                                      cudaWallNodes,
                                                                                                      NumWallNodes,
                                                                                                      min(dx,dy),
                                                                                                      max((x0+FlowNode2D<FP,NUM_COMPONENTS>::dx*TmpMaxX),
                                                                                                      (FlowNode2D<FP,NUM_COMPONENTS>::dy*MaxY)),
                                                                                                      FlowNode2D<FP,NUM_COMPONENTS>::dx,
                                                                                                      FlowNode2D<FP,NUM_COMPONENTS>::dy,
                                                                                                      x0);

                 CUDA_BARRIER((char*)"cuda_SetMinDistanceToWall2D");
                 *o_stream << "OK" << endl;


                 if( TurbExtModel == TEM_Spalart_Allmaras ||
                     TurbExtModel == TEM_vanDriest ||
                     TurbExtModel == TEM_k_eps_Chien ) {

                      *o_stream << "Run cuda_Recalc_y_plus kernel on CUDA device No " << i_gpu << flush;


                      cuda_Recalc_y_plus<<<num_cuda_blocks,num_cuda_threads, 0, cuda_streams[i]>>>(cudaSubDomain,
                                                                                                   TmpMaxX*MaxY,
                                                                                                   cudaWallNodes,
                                                                                                   NumWallNodes,
                                                                                                   min(dx,dy),
                                                                                                   max((x0+FlowNode2D<FP,NUM_COMPONENTS>::dx*TmpMaxX),
                                                                                                       (FlowNode2D<FP,NUM_COMPONENTS>::dy*MaxY)),
                                                                                                   FlowNode2D<FP,NUM_COMPONENTS>::dx,
                                                                                                   FlowNode2D<FP,NUM_COMPONENTS>::dy,
                                                                                                   MaxY);

                      CUDA_BARRIER((char*)"cuda_Recalc_y_plus");
                      *o_stream << "...OK" << endl;
                     }
            }

            *o_stream << "Run cuda_SetInitBoundaryLayer kernel on CUDA device No " << i_gpu << "..." << flush;


            cuda_SetInitBoundaryLayer<<<num_cuda_blocks,num_cuda_threads, 0, cuda_streams[i]>>>(cudaSubDomain,
                                                                                                TmpMaxX*MaxY, iX0, MaxY,
                                                                                                delta_bl,
                                                                                                SigW,SigF,(TurbulenceExtendedModel)TurbExtModel, 
                                                                                                FlowNode2D<FP,NUM_COMPONENTS>::dx,
                                                                                                FlowNode2D<FP,NUM_COMPONENTS>::dy,
                                                                                                cudaHu,
                                                                                                FlowNode2D<FP,NUM_COMPONENTS>::isSrcAdd,
                                                                                                FlowNode2D<FP,NUM_COMPONENTS>::FT,
                                                                                                ProblemType);

            CUDA_BARRIER((char*)"cuda_SetInitBoundaryLayer");
            *o_stream << "OK" << endl;


            CopyDeviceToHost(cudaSubDomain,TmpMatrixPtr,(sizeof(FlowNode2D<FP,NUM_COMPONENTS>))*(TmpMaxX*MaxY));

            iX0 += TmpMaxX;
            TmpDim.SetXY(TmpMaxX,MaxY);
            cudaDimArray->AddElement(&TmpDim);
            o_stream->flush();
            if(MonitorPointsArray) {
                for(int ii_monitor=0;ii_monitor<(int)MonitorPointsArray->GetNumElements();ii_monitor++) {
                        if(MonitorPointsArray->GetElement(ii_monitor).MonitorXY.GetX() >= x0 &&
                           MonitorPointsArray->GetElement(ii_monitor).MonitorXY.GetX() < x0 + FlowNode2D<FP,NUM_COMPONENTS>::dx*TmpMaxX) {
                           MonitorPointsArray->GetElement(ii_monitor).rank = i; 
                        }
                }
            }
        }

        gettimeofday(&mark1,NULL);
        *o_stream << "OK\n" << "Time: " << (FP)(mark1.tv_sec-mark2.tv_sec)+(FP)(mark1.tv_usec-mark2.tv_usec)*1.e-6 << " sec." << endl; 



       DEEPS2D_Run((ofstream*)o_stream,        // ofstream* o_stream,
                   J,                          // UMatrix2D< FlowNode2D< FP,NUM_COMPONENTS > >*
                   C,                          // UMatrix2D< FlowNodeCore2D< FP,NUM_COMPONENTS > >*
                   cudaArraySubDomain,         // UArray< FlowNode2D< FP,NUM_COMPONENTS >* >*
                   cudaArrayCoreSubDomain,     // UArray< FlowNodeCore2D< FP,NUM_COMPONENTS >* >*
                   cudaDimArray,               // UArray< XY<int> >*
                   cudaWallNodesArray,         // UArray< XY<int>* >*
                   cudaCRM2DArray,
                   num_gpus,
                   cuda_streams,
                   cuda_events);

       for (int i = 0; i < num_gpus; i++)  {

           int i_gpu;
           if(isSingleGPU) {
               i_gpu = ActiveSingleGPU;
           } else {
               i_gpu = i;
           }

           if(cudaSetDevice(i_gpu) != cudaSuccess ) {
              *o_stream << "\nError set CUDA device no: "<< i_gpu << endl;
              Exit_OpenHyperFLOW2D(num_gpus);
           }

           cudaState = cudaStreamDestroy(cuda_streams[i]);
           if(cudaState != cudaSuccess ) {
              *o_stream << "\nError destroy stream no: "<< i << endl;
              printf("%s\n", cudaGetErrorString( cudaGetLastError() ) );
              Exit_OpenHyperFLOW2D(num_gpus);
           }

           cudaState = cudaEventDestroy(cuda_events[i]);
           if(cudaState != cudaSuccess ) {
              *o_stream << "\nError destroy event no: "<< i << endl;
              printf("%s\n", cudaGetErrorString( cudaGetLastError() ) );
              Exit_OpenHyperFLOW2D(num_gpus);
           }
       }

       free(cuda_streams);
       free(cuda_events);

       for (int i_dev=0; i_dev < num_gpus;i_dev++)  {

           int i_gpu;
           if(isSingleGPU) {
               i_gpu = ActiveSingleGPU;
           } else {
               i_gpu = i_dev;
           }

           if(cudaSetDevice(i_gpu) != cudaSuccess ) {
              *o_stream << "\nError set CUDA device no: "<< i_gpu << endl;
              Exit_OpenHyperFLOW2D(num_gpus);
           }

#ifdef _P2P_ACCESS_
           if((num_gpus>1)&&(i_dev+1 < num_gpus)){
              // DisableP2PAccess(i_dev,i_dev+1);
           }
#endif // _P2P_ACCESS_

           cudaSubDomain = cudaArraySubDomain->GetElement(i_dev);

           cudaState = cudaFree(cudaSubDomain);

           if(cudaState != cudaSuccess ) {
              *o_stream << "\nError free SubDomain  from GPU memory." << endl;
              Exit_OpenHyperFLOW2D(num_gpus);
           }

           cudaCoreSubDomain = cudaArrayCoreSubDomain->GetElement(i_dev);

           cudaState = cudaFree(cudaCoreSubDomain);

           if(cudaState != cudaSuccess ) {
              *o_stream << "\nError free CoreSubDomain from GPU memory." << endl;
              Exit_OpenHyperFLOW2D(num_gpus);
           }

           if(cudaWallNodesArray->GetNumElements()) {
              cudaWallNodes = cudaWallNodesArray->GetElement(i_dev);

              cudaState = cudaFree(cudaWallNodes);

              if(cudaState != cudaSuccess ) {
                 *o_stream << "\nError free WallNodes array from GPU memory." << endl;
                 Exit_OpenHyperFLOW2D(num_gpus);
              }
           }
#ifdef _DEVICE_MMAP_
           dt_min_host = dt_min_host_Array->GetElement(i_dev);

           cudaState = cudaFreeHost(dt_min_host);
           if(cudaState != cudaSuccess ) {
              *o_stream << "\nError free dt_min from GPU memory." << endl;
              Exit_OpenHyperFLOW2D(num_gpus);
           }
#else
           dt_min_device = dt_min_device_Array->GetElement(i_dev);

           cudaState = cudaFree(dt_min_device);

           if(cudaState != cudaSuccess ) {
              *o_stream << "\nError free dt_min from GPU memory." << endl;
              Exit_OpenHyperFLOW2D(num_gpus);
           }
#endif //_DEVICE_MMAP_

           cudaHu = cudaHuArray->GetElement(i_dev);

           cudaState = cudaFree(cudaHu);

           if(cudaState != cudaSuccess ) {
              *o_stream << "\nError free cudaHu from GPU memory." << endl;
              Exit_OpenHyperFLOW2D(num_gpus);
           }
       }
       //>>>>>>>>>>>>>>>>>>>>>>
       //DataSnapshot(OutFileName,WM_REWRITE);
       //>>>>>>>>>>>>>>>>>>>>>>

       delete cudaArraySubDomain;
       delete cudaArrayCoreSubDomain;
       delete cudaDimArray;
       delete cudaHuArray;
       delete dt_min_host_Array;
       delete dt_min_device_Array;
       delete cudaWallNodesArray;

#ifdef _DEBUG_0
    }__except(SysException e) {{
            const char ConstErrorMessage[]=" handled by <LibExcept> module in <InputData>.\n";
            *o_stream << "SIG";
            if (e == SIGSEGV)
                *o_stream << "SEGV" << ConstErrorMessage ;
            else if (e == SIGBUS)
                *o_stream << "BUS" << ConstErrorMessage ;
            else if (e == SIGFPE)
                *o_stream << "FPE" << ConstErrorMessage ;
            else if (e == SIGINT)
                *o_stream << "INT" << ConstErrorMessage ;
            else if (e == SIGABRT)
                *o_stream << "ABRT" << ConstErrorMessage ;
            else if (e == SIGIO)
                *o_stream << "IO" << ConstErrorMessage ;
            else if (e == SIGTRAP)
                *o_stream << "TRAP" << ConstErrorMessage ;
            else
                *o_stream << " No: "<< e << ConstErrorMessage ;
        }
    } __end_except;
#endif  // _DEBUG_0
        *o_stream << "Computation stopped.\n";
        o_stream->flush();
        Exit_OpenHyperFLOW2D(num_gpus);
        return 0;
}
#endif //_CUDA_

